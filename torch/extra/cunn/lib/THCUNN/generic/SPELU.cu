
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SPELU.cu"
#else

void THNN_(SPELU_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           THCTensor *weight,
           long nOutputPlane)
{
  THCTensor_(resizeAs)(state, output, input);

  real *w = THCTensor_(data)(state, weight);

  if (nOutputPlane == 0)
  {
    THC_pointwiseApply2(state, output, input, SPELUUpdateOutput<real>(w));
  }
  else
  {
    int ndim = THCTensor_(nDimension)(state, input);
    input = THCTensor_(newContiguous)(state, input);

    int n = THCTensor_(nElement)(state, input);
    int mapSize = 1;
    if (ndim == 3)
      mapSize = (input->size[1] * input->size[2]);
    else if (ndim == 4)
      mapSize = (input->size[2] * input->size[3]);
    int nElemsPerSample = nOutputPlane * mapSize;
    speluForward<<<GET_BLOCKS(n), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
      THCTensor_(data)(state, output),
      THCTensor_(data)(state, input),
      w,
      n, nElemsPerSample, mapSize
    );
    THCudaCheck(cudaGetLastError());
    THCTensor_(free)(state, input);
  }
}

void THNN_(SPELU_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCTensor *weight,
           long nOutputPlane)
{
  THCUNN_check_nElement(state, input, gradOutput);
  THCTensor_(resizeAs)(state, gradInput, input);

  real *w = THCTensor_(data)(state, weight);
  if (nOutputPlane == 0)
  {
    THC_pointwiseApply3(state, gradInput, gradOutput, input, SPELUUpdateGradInput<real>(w));
  }
  else
  {
    int ndim = THCTensor_(nDimension)(state, input);
    input = THCTensor_(newContiguous)(state, input);
    gradOutput = THCTensor_(newContiguous)(state, gradOutput);

    int n = THCTensor_(nElement)(state, input);
    int mapSize = 1;
    if (ndim == 3)
      mapSize = (input->size[1] * input->size[2]);
    else if (ndim == 4)
      mapSize = (input->size[2] * input->size[3]);
    int nElemsPerSample = nOutputPlane * mapSize;
    speluBackward<<<GET_BLOCKS(n), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
      THCTensor_(data)(state, gradInput),
      THCTensor_(data)(state, input),
      w,
      THCTensor_(data)(state, gradOutput),
      n, nElemsPerSample, mapSize
    );
    THCudaCheck(cudaGetLastError());
    THCTensor_(free)(state, input);
    THCTensor_(free)(state, gradOutput);
  }
}

void THNN_(SPELU_accGradParameters)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCTensor *weight,
           THCTensor *gradWeight,
           THCTensor *gradWeightBuf,
           THCTensor *gradWeightBuf2,
           long nOutputPlane,
           real scale)
{
  THCUNN_check_nElement(state, input, gradOutput);
  // use grad input for temporary storage, then call updateGradInput again

  if (nOutputPlane == 0)
  {
    THC_pointwiseApply3(state, gradInput, input, gradOutput, SPELUAccGradParametersShared<real>());

    // introduces a sync point
    real sum = ScalarConvert<accreal, real>::to(THCTensor_(sumall)(state, gradInput));
    real w = THCTensor_(get1d)(state, gradWeight, 0);
    THCTensor_(set1d)(state, gradWeight, 0, w + sum * scale);

    // restore gradInput
    THNN_(SPELU_updateGradInput)(state, input, gradOutput, gradInput, weight, nOutputPlane);
  }
  else
  {
    int ndim = THCTensor_(nDimension)(state, input);

    if (ndim == 1)
    {
      THC_pointwiseApply3(state, gradWeight, input, gradOutput, SPELUAccGradParameters1to1<real>(scale));
    }
    else
    {
      THC_pointwiseApply3(state, gradInput, input, gradOutput, SPELUAccGradParameters<real>(scale));
      THCTensor *sumbuf = gradWeightBuf2;
      THCTensor_(resizeAs)(state, gradWeightBuf, gradWeight);

      if (ndim == 2)
      {
        THCTensor_(sum)(state, gradWeightBuf, gradInput, 0);
        THCTensor_(cadd)(state, gradWeight, gradWeight, scale, gradWeightBuf);
      }
      else if (ndim == 3)
      {
        THCTensor *buffer = THCTensor_(newContiguous)(state, gradInput);
        THCTensor_(resize2d)(state, buffer, nOutputPlane, input->size[1] * input->size[2]);
        THCTensor_(sum)(state, gradWeightBuf, buffer, 1);
        THCTensor_(cadd)(state, gradWeight, gradWeight, scale, gradWeightBuf);
        THCTensor_(free)(state, buffer);
      }
      else if (ndim == 4)
      {
        THCTensor *buffer = THCTensor_(newContiguous)(state, gradInput);
        THCTensor_(resize3d)(state, buffer, input->size[0], nOutputPlane, input->size[2] * input->size[3]);
        THCTensor_(resize2d)(state, sumbuf, input->size[0], nOutputPlane);
        THCTensor_(sum)(state, sumbuf, buffer, 2);
        THCTensor_(sum)(state, gradWeightBuf, sumbuf, 0);
        THCTensor_(cadd)(state, gradWeight, gradWeight, scale, gradWeightBuf);
        THCTensor_(free)(state, buffer);
      }

      // restore gradInput
      THNN_(SPELU_updateGradInput)(state, input, gradOutput, gradInput, weight, nOutputPlane);
    }
  }
}

#endif
